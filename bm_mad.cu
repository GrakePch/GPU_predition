#include <stdio.h>
#include <hip/hip_runtime.h>

// Define the number of iterations
#define N 1e7

// Kernel for testing mad.f32
__global__ void microbenchmark(float *input1, float *input2, float *output) {
    float temp = *input1;

    for (int i = 0; i < N; i++) {
        temp = temp * (*input2) + (*input2); // mad.f32 computation
    }
    *output = temp;
}

int main(int argc, char *argv[]) {
    // Ensure correct input arguments
    if (argc != 3) {
        printf("Usage: <freq in GHz> <GPU id>\n");
        return 1;
    }

    // Parse input arguments
    float freq_ghz = atof(argv[1]); // GPU frequency in GHz
    int gpu_id = atoi(argv[2]);    // GPU device ID
    printf("Selected GPU ID: %d\n", gpu_id);
    printf("GPU frequency: %f GHz\n", freq_ghz);

    // Set the selected GPU device
    hipSetDevice(gpu_id);

    // Host and device variables
    float *d_input1, *d_input2, *d_output;
    float h_input1 = 1.0f, h_input2 = 1.234f, h_output;

    // Allocate device memory
    hipMalloc(&d_input1, sizeof(float));
    hipMalloc(&d_input2, sizeof(float));
    hipMalloc(&d_output, sizeof(float));

    // Copy input values to device
    hipMemcpy(d_input1, &h_input1, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, &h_input2, sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch the kernel and record time
    hipEventRecord(start);
    microbenchmark<<<1, 1>>>(d_input1, d_input2, d_output); // Single thread
    hipEventRecord(stop);

    // Wait for the kernel to complete and synchronize
    hipEventSynchronize(stop);

    // Get elapsed time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Free memory and destroy events
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Print results
    printf("Time elapsed: %f ms\n", milliseconds);
    float cycles = (milliseconds / 1000.0) * freq_ghz * 1e9 / N; // Convert to cycles
    printf("mad.f32 delay: %f cycles\n", cycles);

    return 0;
}
