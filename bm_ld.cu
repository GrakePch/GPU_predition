#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (64 * 1024 * 1024)  // array length

__global__ void measure_memory_load_cycles(volatile int *data, int *index, unsigned long long *cycle_diff) {
    int idx = 0;

    unsigned long long start = clock64();

    idx = index[idx];  // Pointer chasing

    unsigned long long end = clock64();

    *cycle_diff = end - start;

    // Prevent compiler optimization
    data[0] = idx;
}

int main() {
    
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    hipSetDevice(deviceCount-1);

    int *h_index;
    int *d_index, *d_data;
    unsigned long long *d_cycle_diff;
    unsigned long long cycle_diff;

    h_index = (int *)malloc(N * sizeof(int));

    for (int i = 0; i < N; i++) {
        h_index[i] = (i * 37) % N;  // Randomize access pattern
    }

    hipMalloc(&d_index, N * sizeof(int));
    hipMalloc(&d_data, sizeof(int));
    hipMalloc(&d_cycle_diff, sizeof(unsigned long long));

    // Load from global mem
    hipMemcpy(d_index, h_index, N * sizeof(int), hipMemcpyHostToDevice);
    measure_memory_load_cycles<<<1, 1>>>(d_data, d_index, d_cycle_diff);
    hipMemcpy(&cycle_diff, d_cycle_diff, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    printf("Global memory load latency: %llu cycles\n", cycle_diff);

    hipFree(d_index);
    hipFree(d_data);
    hipFree(d_cycle_diff);
    free(h_index);

    return 0;
}
