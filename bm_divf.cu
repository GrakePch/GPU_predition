#include <stdio.h>
#include <hip/hip_runtime.h>

# define N 1e7

__global__ void microbenchmark(float *input, float *output) {
    float temp = *input;

    for (int i = 0; i < N; i++) {
        temp /= 1.234f; // Example computation
    }
    *output = temp;
}

int main(int argc, char *argv[]) {
    
    if (argc != 3) {
        printf("usage: <freq in GHz> <GPU id>\n");
        exit(1);
    }

    
    float f = atof(argv[1]);
    printf("GPU frequency %f Ghz.\n", f);

    float *d_input, *d_output;
    float h_input = 1.0f, h_output;

    hipSetDevice(atoi(argv[2]));

    hipMalloc(&d_input, sizeof(float));
    hipMalloc(&d_output, sizeof(float));

    hipMemcpy(d_input, &h_input, sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    microbenchmark<<<1, 1>>>(d_input, d_output);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Time elapsed: %f ms\n", milliseconds);
    float cycles = milliseconds / 1000 / N * f * 1e9;
    printf("Cycle: %f\n", cycles);
    return 0;
}
