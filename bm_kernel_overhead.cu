#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void emptyKernel() {
}

int main() {
    hipEvent_t start, stop;
    float elapsedTime;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threadsPerBlock = 256; 
    int blocks;

    FILE *outputFile = fopen("kernel_launch_overhead.csv", "w");
    if (!outputFile) {
        fprintf(stderr, "Failed to open output file.\n");
        return -1;
    }
    fprintf(outputFile, "NumThreads,Overhead(s)\n");

    printf("Measuring kernel launch overhead...\n");
    
    // Warm-up
    emptyKernel<<<1, threadsPerBlock>>>();
    hipDeviceSynchronize();

    int iterations = 10;

    for (long long numThreads = 1e6; numThreads <= 8e6; numThreads += 1e6) {
        
        blocks = (numThreads + threadsPerBlock - 1) / threadsPerBlock;

        float avgOverhead = 0;
        for (int i = 0; i < iterations ;++i)
        {
            hipEventRecord(start);
            emptyKernel<<<blocks, threadsPerBlock>>>();
            hipEventRecord(stop);
            hipEventSynchronize(stop);

            hipEventElapsedTime(&elapsedTime, start, stop); // milliseconds

            avgOverhead += elapsedTime / 1000.0;
        }

        avgOverhead /= iterations;

        printf("Threads: %lld, Overhead: %.8f seconds\n", numThreads, avgOverhead);
        fprintf(outputFile, "%lld,%.8f\n", numThreads, avgOverhead);
    }

    for (long long numThreads = 1000; numThreads <= 1e7; numThreads *= 10) {
        
        blocks = (numThreads + threadsPerBlock - 1) / threadsPerBlock;

        float avgOverhead = 0;
        for (int i = 0; i < iterations ;++i)
        {
            hipEventRecord(start);
            emptyKernel<<<blocks, threadsPerBlock>>>();
            hipEventRecord(stop);
            hipEventSynchronize(stop);

            hipEventElapsedTime(&elapsedTime, start, stop); // milliseconds

            avgOverhead += elapsedTime / 1000.0;
        }

        avgOverhead /= iterations;

        printf("Threads: %lld, Overhead: %.8f seconds\n", numThreads, avgOverhead);
        fprintf(outputFile, "%lld,%.8f\n", numThreads, avgOverhead);
    }

    fclose(outputFile);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Results saved to 'kernel_launch_overhead.csv'.\n");
    return 0;
}