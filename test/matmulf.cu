#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixMulKernel(float* A, float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  
    int col = blockIdx.x * blockDim.x + threadIdx.x; 

    if (row < n && col < n) {
        float value = 0.0f;
        for (int k = 0; k < n; k++) {
            value += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = value;
    }
}



int main(int argc, char * argv[]) {
    int n;
    
    if(argc != 3)
    {
        fprintf(stderr, "need arguments: matDim gpuId\n");
        exit(1);
    }

    n = (unsigned int) atoi(argv[1]);
    hipSetDevice(atoi(argv[2]));

    size_t bytes = n * n * sizeof(float);

    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);

    for (int i = 0; i < n * n; i++) {
        h_A[i] = (float)rand() / RAND_MAX;
        h_B[i] = (float)rand() / RAND_MAX;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int it = 20;
    float milliseconds = 0;

    
    for (int i = 0; i < it; ++i) {

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEventRecord(start);
    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, n);
    hipEventRecord(stop);

    
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n", ms/1e3);
    if (i > 0)  // use first run as warm-up to increase accuracy
        milliseconds += ms;
    }
    
    milliseconds /= it - 1;

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    
    printf("GPU time = %lf secs\n", milliseconds / 1e3);

    
    hipDeviceSynchronize();  // block host till device is done.
    
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);




    return 0;
}
