#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(float *, float *, float, int);

int main(int argc, char *argv[]) {
    int i;
    int num = 0;        // number of elements in the arrays
    float *a, *c;     // arrays at host
    float *ad, *cd;  // arrays at device

    if (argc != 3) {
        printf("need arguments: numelements gpuId\n");
        exit(1);
    }

    num = atoi(argv[1]);
    hipSetDevice(atoi(argv[2]));

    a = (float *)malloc(num * sizeof(float));
    if (!a) {
        printf("Cannot allocate array a with %d elements\n", num);
        exit(1);
    }

    c = (float *)malloc(num * sizeof(float));
    if (!c) {
        printf("Cannot allocate array c with %d elements\n", num);
        exit(1);
    }

    // Fill out arrays a and b with some random numbers
    srand(time(0));
    for (i = 0; i < num; i++) {
        a[i] = rand() % num;
    }

    // Now zero C[] in preparation for GPU version
    for (i = 0; i < num; i++)
        c[i] = 0;

    int numblocks;
    int threadsperblock = 256;

    if ((num % threadsperblock) == 0)
        numblocks = num / threadsperblock;
    else
        numblocks = (num / threadsperblock) > 0 ? (num / threadsperblock) + 1 : 1;

    printf("GPU: %d blocks of %d threads each\n", numblocks, threadsperblock);

    // assume a block can have THREADS threads
    dim3 grid(numblocks, 1, 1);
    dim3 block(threadsperblock, 1, 1);

    hipMalloc((void **)&ad, num * sizeof(float));
    if (!ad) {
        printf("cannot allocated array ad of %d elements\n", num);
        exit(1);
    }

    hipMalloc((void **)&cd, num * sizeof(float));
    if (!cd) {
        printf("cannot allocated array cd of %d elements\n", num);
        exit(1);
    }

    // mov a and b to the device
    hipMemcpy(ad, a, num * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch the kernel
    hipEventRecord(start);
    kernel<<<numblocks, threadsperblock>>>(ad, cd, 1.234, num);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // bring data back
    hipMemcpy(c, cd, num * sizeof(float), hipMemcpyDeviceToHost);

    printf("GPU time = %lf secs\n", milliseconds / 1e3);

    hipDeviceSynchronize();  // block host till device is done.

    free(a);
    free(c);

    hipFree(ad);
    hipFree(cd);
}

__global__ void kernel(float *a, float *c, float d, int n) {
    int index;

    index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < n)
        c[index] = a[index] * d;
}
