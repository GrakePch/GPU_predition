#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void microbenchmarkAdd(float *input, float *output, int N) {
    // float temp = *input;
    // for (int i = 0; i < N; i++) {
    //     temp /= 1.345f; // Example computation
    // }
    // *output = temp;
}

int main(int argc, char *argv[]) {
    
    if (argc != 2) {
        printf("usage:  gpu_frequency in Ghz\n");
        exit(1);
    }

    
    float f = atof(argv[1]);
    printf("GPU frequency %f Ghz.\n", f);

    const int N = 1e7; // Number of iterations
    float *d_input, *d_output;
    float h_input = 1.0f, h_output;

    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    hipSetDevice(deviceCount-1);

    // Allocate device memory
    hipMalloc(&d_input, sizeof(float));
    hipMalloc(&d_output, sizeof(float));

    // Copy input to device
    hipMemcpy(d_input, &h_input, sizeof(float), hipMemcpyHostToDevice);

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // Launch the kernel
    microbenchmarkAdd<<<1024000, 1024>>>(d_input, d_output, N);

    // Record the stop event
    hipEventRecord(stop);

    // Wait for the event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy the result back
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Free memory and destroy events
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Time elapsed: %f ms\n", milliseconds);
    float cycles = milliseconds / 1000 / N * f * 1e9;
    printf("Cycle: %f\n", cycles);
    return 0;
}
