#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void microbenchmarkAdd(float *input, float *output, int N) {
    float temp = *input;
    for (int i = 0; i < N; i++) {
        temp += 1.345f; // Example computation
    }
    *output = temp;
}

int main(int argc, char *argv[]) {
    
    if (argc != 2) {
        printf("usage:  gpu_frequency in Ghz\n");
        exit(1);
    }

    
    float f = atof(argv[1]);
    printf("GPU frequency %f Ghz.\n", f);

    const int N = 1e7; // Number of iterations
    float *d_input, *d_output;
    float h_input = 1.0f, h_output;

    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    hipSetDevice(deviceCount-1);

    hipMalloc(&d_input, sizeof(float));
    hipMalloc(&d_output, sizeof(float));

    hipMemcpy(d_input, &h_input, sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    microbenchmarkAdd<<<1, 1>>>(d_input, d_output, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Time elapsed: %f ms\n", milliseconds);
    float cycles = milliseconds / 1000 / N * f * 1e9;
    printf("Cycle: %f\n", cycles);
    return 0;
}
