#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define RANGE 17.78

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void multVector(float *a, float *b, float *c, int n, int opPerTh);

/**** end of the kernel declaration ***/

int main(int argc, char *argv[]) {
    int n = 0;            // number of elements in the arrays
    int i;                // loop index
    float *a, *b, *c;     // The arrays that will be processed in the host.
    float *temp;          // array in host used in the sequential code.
    float *ad, *bd, *cd;  // The arrays that will be processed in the device.
    clock_t start, end;   // to meaure the time taken by a specific part of code

    if (argc != 2) {
        printf("usage:  ./vectorprog n\n");
        printf("n = number of elements in each vector\n");
        exit(1);
    }

    n = atoi(argv[1]);
    printf("Each vector will have %d elements\n", n);

    // Allocating the arrays in the host

    if (!(a = (float *)malloc(n * sizeof(float)))) {
        printf("Error allocating array a\n");
        exit(1);
    }

    if (!(b = (float *)malloc(n * sizeof(float)))) {
        printf("Error allocating array b\n");
        exit(1);
    }

    if (!(c = (float *)malloc(n * sizeof(float)))) {
        printf("Error allocating array c\n");
        exit(1);
    }

    if (!(temp = (float *)malloc(n * sizeof(float)))) {
        printf("Error allocating array temp\n");
        exit(1);
    }

    // Fill out the arrays with random numbers between 0 and RANGE;
    srand((unsigned int)time(NULL));
    for (i = 0; i < n; i++) {
        a[i] = ((float)rand() / (float)(RAND_MAX)) * RANGE;
        b[i] = ((float)rand() / (float)(RAND_MAX)) * RANGE;
        c[i] = ((float)rand() / (float)(RAND_MAX)) * RANGE;
        temp[i] = c[i];  // temp is just another copy of C
    }

    // The sequential part
    start = clock();
    for (i = 0; i < n; i++)
        temp[i] += a[i] * b[i];
    end = clock();
    printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
    // The GPU part

    /* TODO: in this part you need to do the following:
            1. allocate ad, bd, and cd in the device
            2. send a, b, and c to the device
            */
    int numBlocks = 4;
    int threadsPerBlock = 500;
    int opPerThread;
    if ((n % (numBlocks * threadsPerBlock)) == 0)
        opPerThread = n / (numBlocks * threadsPerBlock);
    else
        opPerThread = n / (numBlocks * threadsPerBlock) + 1;

    start = clock();

    hipMalloc((void **)&ad, n * sizeof(float));
    if (!ad) {
        printf("Error allocating array ad\n");
        exit(1);
    }
    hipMalloc((void **)&bd, n * sizeof(float));
    if (!bd) {
        printf("Error allocating array bd\n");
        exit(1);
    }
    hipMalloc((void **)&cd, n * sizeof(float));
    if (!cd) {
        printf("Error allocating array cd\n");
        exit(1);
    }

    hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bd, b, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cd, c, n * sizeof(float), hipMemcpyHostToDevice);


    /* TODO:
            3. write the kernel, call it: vecGPU
            4. call the kernel (the kernel itself will be written at the comment at the end of this file),
               you need to decide about the number of threads, blocks, etc and their geometry.
            */
    multVector<<<numBlocks, threadsPerBlock>>>(ad, bd, cd, n, opPerThread);
    /* TODO:
            5. bring the cd array back from the device and store it in c array (declared earlier in main)
            6. free ad, bd, and cd
    */
    hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
    
    end = clock();

    printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
    /******************  The end of the GPU part: Do not modify anything in main() below this line  ************/

    // checking the correctness of the GPU part
    for (i = 0; i < n; i++)
        if (fabs(temp[i] - c[i]) >= 0.009)  // compare up to the second degit in floating point
            printf("Element %d in the result array does not match the sequential version\n", i);

    // Free the arrays in the host
    free(a);
    free(b);
    free(c);
    free(temp);

    return 0;
}

/**** TODO: Write the kernel itself below this line *****/
__global__ void multVector(float *a, float *b, float *c, int n, int opPerTh) {
    int index;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int op = 0; op < opPerTh; ++op) {
        index = tid * opPerTh + op;
        if (index < n)
            c[index] += a[index] * b[index];
    }
}
