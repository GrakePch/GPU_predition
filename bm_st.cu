#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (64 * 1024 * 1024)  // array length

__global__ void measure_memory_store_cycles(volatile int *data, int value, unsigned long long *cycle_diff) {
    unsigned long long start = clock64();

    data[0] = value;

    unsigned long long end = clock64();
    *cycle_diff = end - start;
}

int main() {
    
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    hipSetDevice(deviceCount-1);

    int *h_index;
    int *d_index, *d_data;
    unsigned long long *d_cycle_diff;
    unsigned long long cycle_diff;

    h_index = (int *)malloc(N * sizeof(int));

    for (int i = 0; i < N; i++) {
        h_index[i] = (i * 37) % N;  // Randomize access pattern
    }

    hipMalloc(&d_index, N * sizeof(int));
    hipMalloc(&d_data, sizeof(int));
    hipMalloc(&d_cycle_diff, sizeof(unsigned long long));

    // Store to global mem
    hipMemcpy(d_index, h_index, N * sizeof(int), hipMemcpyHostToDevice);
    measure_memory_store_cycles<<<1, 1>>>(d_data, 2, d_cycle_diff);
    hipMemcpy(&cycle_diff, d_cycle_diff, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    printf("Global memory store latency: %llu cycles\n", cycle_diff);

    hipFree(d_index);
    hipFree(d_data);
    hipFree(d_cycle_diff);
    free(h_index);

    return 0;
}
