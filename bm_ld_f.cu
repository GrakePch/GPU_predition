#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (64 * 1024 * 1024)  // array length

__global__ void measure_memory_load_cycles(volatile float *data, float *index, unsigned long long *cycle_diff) {
    int idx = 0;

    unsigned long long start = clock64();

    idx = (int)index[idx];  // Pointer chasing

    unsigned long long end = clock64();

    *cycle_diff = end - start;

    // Prevent compiler optimization
    data[0] = idx;
}

int main() {
    
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    hipSetDevice(deviceCount-1);

    float *h_index;
    float *d_index, *d_data;
    unsigned long long *d_cycle_diff;
    unsigned long long cycle_diff;

    h_index = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        h_index[i] = (i * 37) % N;  // Randomize access pattern
    }

    hipMalloc(&d_index, N * sizeof(float));
    hipMalloc(&d_data, sizeof(float));
    hipMalloc(&d_cycle_diff, sizeof(unsigned long long));

    // Load from global mem
    hipMemcpy(d_index, h_index, N * sizeof(float), hipMemcpyHostToDevice);
    measure_memory_load_cycles<<<1, 1>>>(d_data, d_index, d_cycle_diff);
    hipMemcpy(&cycle_diff, d_cycle_diff, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    printf("Global memory load latency: %llu cycles\n", cycle_diff);

    hipFree(d_index);
    hipFree(d_data);
    hipFree(d_cycle_diff);
    free(h_index);

    return 0;
}
