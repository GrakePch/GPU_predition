#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void memoryBandwidthKernel(float *dst, const float *src, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        dst[idx] = src[idx];
    }
}

void testBandwidth(size_t size_in_mb) {
    size_t size = size_in_mb * 1024 * 1024; // Convert MB to bytes
    size_t num_elements = size / sizeof(float);

    float *h_src = (float *)malloc(size);
    float *h_dst = (float *)malloc(size);
    if (!h_src || !h_dst) {
        fprintf(stderr, "Host memory allocation failed\n");
        exit(EXIT_FAILURE);
    }

    for (size_t i = 0; i < num_elements; ++i) {
        h_src[i] = (float)i;
    }

    float *d_src, *d_dst;
    hipMalloc((void **)&d_src, size);
    hipMalloc((void **)&d_dst, size);

    hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int blocks_per_grid = (num_elements + threads_per_block - 1) / threads_per_block;

    // Warm-up run
    // To avoid initialization overhead, and get some cache.
    memoryBandwidthKernel<<<blocks_per_grid, threads_per_block>>>(d_dst, d_src, num_elements);
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    memoryBandwidthKernel<<<blocks_per_grid, threads_per_block>>>(d_dst, d_src, num_elements);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed_ms = 0.0f;
    hipEventElapsedTime(&elapsed_ms, start, stop);

    float bandwidth = (2.0f * size) / (elapsed_ms * 1e6); // GB/s
    printf("Bandwidth: %.2f GB/s\n", bandwidth);

    free(h_src);
    free(h_dst);
    hipFree(d_src);
    hipFree(d_dst);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    size_t test_size_mb = 1024; // Test with 1GB of data
    testBandwidth(test_size_mb);
    return 0;
}

