#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA(call) do {                                   \
    hipError_t err = call;                                     \
    if (err != hipSuccess) {                                   \
        fprintf(stderr, "CUDA error at %s:%d: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(err));   \
        exit(EXIT_FAILURE);                                     \
    }                                                           \
} while (0)

__global__ void memoryBandwidthKernel(float *dst, const float *src, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        dst[idx] = src[idx];
    }
}

void testBandwidth(size_t size_in_mb) {
    size_t size = size_in_mb * 1024 * 1024; // Convert MB to bytes
    size_t num_elements = size / sizeof(float);

    float *h_src = (float *)malloc(size);
    float *h_dst = (float *)malloc(size);
    if (!h_src || !h_dst) {
        fprintf(stderr, "Host memory allocation failed\n");
        exit(EXIT_FAILURE);
    }

    for (size_t i = 0; i < num_elements; ++i) {
        h_src[i] = (float)i;
    }

    float *d_src, *d_dst;
    CHECK_CUDA(hipMalloc((void **)&d_src, size));
    CHECK_CUDA(hipMalloc((void **)&d_dst, size));

    CHECK_CUDA(hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice));

    int threads_per_block = 256;
    int blocks_per_grid = (num_elements + threads_per_block - 1) / threads_per_block;

    // Warm-up run
    // To avoid initialization overhead, and get some cache.
    memoryBandwidthKernel<<<blocks_per_grid, threads_per_block>>>(d_dst, d_src, num_elements);
    CHECK_CUDA(hipDeviceSynchronize());

    // Measure kernel execution time
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    memoryBandwidthKernel<<<blocks_per_grid, threads_per_block>>>(d_dst, d_src, num_elements);

    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    float elapsed_ms = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&elapsed_ms, start, stop));

    // Calculate bandwidth
    float bandwidth = (2.0f * size) / (elapsed_ms * 1e6); // GB/s
    printf("Bandwidth: %.2f GB/s\n", bandwidth);

    free(h_src);
    free(h_dst);
    CHECK_CUDA(hipFree(d_src));
    CHECK_CUDA(hipFree(d_dst));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
}

int main() {
    size_t test_size_mb = 1024; // Test with 1GB of data
    testBandwidth(test_size_mb);
    return 0;
}

