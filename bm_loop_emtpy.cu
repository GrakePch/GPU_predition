#include <stdio.h>
#include <hip/hip_runtime.h>

# define N 1e7

__global__ void microbenchmark(float *input, float *output) {
    float temp = *input;
    
    #pragma unroll 1
    for (int i = 0; i < N; i++) {
    }
    *output = temp;
}

int main(int argc, char *argv[]) {
    
    if (argc != 2) {
        printf("usage:  gpu_frequency in Ghz\n");
        exit(1);
    }

    
    float f = atof(argv[1]);
    printf("GPU frequency %f Ghz.\n", f);

    float *d_input, *d_output;
    float h_input = 1.0f, h_output;

    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    hipSetDevice(deviceCount-1);

    hipMalloc(&d_input, sizeof(float));
    hipMalloc(&d_output, sizeof(float));

    hipMemcpy(d_input, &h_input, sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    microbenchmark<<<1, 1>>>(d_input, d_output);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Time elapsed: %f ms\n", milliseconds);
    float cycles = milliseconds / 1000 / N * f * 1e9;
    printf("Cycle: %f\n", cycles);
    return 0;
}