#include <stdio.h>
#include <hip/hip_runtime.h>

// Define the number of iterations
#define N 1e7 

// Kernel for testing bitwise AND operation
__global__ void microbenchmark(unsigned int *input, unsigned int *output) {
    unsigned int temp = *input;
    for (int i = 0; i < N; i++) {
        temp = (temp & 0xFFFF) ^ 0x1234;  // Add more operations for complexity
    }
    *output = temp;
}

int main(int argc, char *argv[]) {
    // Ensure correct input arguments
    if (argc != 3) {
        printf("Usage: <freq in GHz> <GPU id>\n");
        return 1;
    }

    // Parse input arguments
    float freq_ghz = atof(argv[1]); // GPU frequency in GHz
    int gpu_id = atoi(argv[2]);    // GPU ID
    printf("Selected GPU ID: %d\n", gpu_id);
    printf("GPU frequency: %f GHz\n", freq_ghz);

    // Set the selected GPU device
    hipSetDevice(gpu_id);

    // Host and device variables
    unsigned int *d_input, *d_output;
    unsigned int h_input = 0xFFFFFFFF, h_output;

    // Allocate device memory
    hipMalloc(&d_input, sizeof(unsigned int));
    hipMalloc(&d_output, sizeof(unsigned int));

    // Copy input value to device
    hipMemcpy(d_input, &h_input, sizeof(unsigned int), hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch the kernel and record time
    hipEventRecord(start);
    microbenchmark<<<1, 1>>>(d_input, d_output); // Single-thread kernel
    hipEventRecord(stop);

    // Wait for the kernel to complete and synchronize
    hipEventSynchronize(stop);

    // Get elapsed time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    hipMemcpy(&h_output, d_output, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Free memory and destroy events
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Print results
    printf("Time elapsed: %f ms\n", milliseconds);
    float cycles = (milliseconds / 1000.0) * freq_ghz * 1e9 / N; // Convert to cycles
    printf("and delay: %f cycles\n", cycles);

    return 0;
}
